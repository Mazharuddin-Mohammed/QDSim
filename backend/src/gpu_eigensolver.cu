/**
 * @file gpu_eigensolver.cu
 * @brief CUDA implementation of eigenvalue solver for Schrödinger equation.
 *
 * This file contains the CUDA implementation of the eigenvalue solver for the
 * Schrödinger equation using cuSOLVER and cuSPARSE libraries.
 *
 * Author: Dr. Mazharuddin Mohammed
 */

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <thrust/complex.h>
#include <complex>
#include <vector>
#include <iostream>

// Constants
const double TOLERANCE = 1e-10;
const int MAX_ITERATIONS = 1000;

/**
 * @brief Converts a sparse matrix from CSR format to dense format.
 *
 * @param handle The cuSPARSE handle
 * @param rows The number of rows in the matrix
 * @param cols The number of columns in the matrix
 * @param nnz The number of non-zero elements in the matrix
 * @param csrRowPtr The CSR row pointer array
 * @param csrColInd The CSR column index array
 * @param csrVal The CSR value array
 * @param dense The dense matrix (output)
 * @return hipError_t The CUDA error code
 */
hipError_t csr2dense(
    hipsparseHandle_t handle,
    int rows,
    int cols,
    int nnz,
    const int* csrRowPtr,
    const int* csrColInd,
    const thrust::complex<double>* csrVal,
    thrust::complex<double>* dense) {

    // Create matrix descriptors
    hipsparseMatDescr_t descr = 0;
    hipsparseStatus_t status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        return hipErrorInvalidValue;
    }

    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    // Convert CSR to dense
    // Use cusparseXcsr2dense for CUDA 11.0+
#if CUDART_VERSION >= 11000
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB;
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    // Create sparse matrix A in CSR format
    status = hipsparseCreateCsr(&matA, rows, cols, nnz,
                              (void*)csrRowPtr, (void*)csrColInd, (void*)csrVal,
                              HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                              HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        return hipErrorInvalidValue;
    }

    // Create dense matrix B
    status = hipsparseCreateDnMat(&matB, rows, cols, rows, dense, HIP_C_64F, HIPSPARSE_ORDER_COL);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        hipsparseDestroySpMat(matA);
        return hipErrorInvalidValue;
    }

    // Get buffer size
    status = hipsparseSparseToDense_bufferSize(handle, matA, matB, HIPSPARSE_SPARSETODENSE_ALG_DEFAULT, &bufferSize);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        hipsparseDestroyDnMat(matB);
        hipsparseDestroySpMat(matA);
        return hipErrorInvalidValue;
    }

    // Allocate buffer
    hipError_t cuda_status = hipMalloc(&dBuffer, bufferSize);
    if (cuda_status != hipSuccess) {
        hipsparseDestroyDnMat(matB);
        hipsparseDestroySpMat(matA);
        return cuda_status;
    }

    // Execute sparse to dense conversion
    status = hipsparseSparseToDense(handle, matA, matB, HIPSPARSE_SPARSETODENSE_ALG_DEFAULT, dBuffer);

    // Clean up
    hipFree(dBuffer);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matA);
#else
    // For older CUDA versions, use deprecated function
    status = hipsparseZcsr2dense(
        handle,
        rows,
        cols,
        descr,
        reinterpret_cast<const hipDoubleComplex*>(csrVal),
        csrRowPtr,
        csrColInd,
        reinterpret_cast<hipDoubleComplex*>(dense),
        rows
    );
#endif

    hipsparseDestroyMatDescr(descr);

    if (status != HIPSPARSE_STATUS_SUCCESS) {
        return hipErrorInvalidValue;
    }

    return hipSuccess;
}

/**
 * @brief Solves the generalized eigenvalue problem Ax = λBx using cuSOLVER.
 *
 * @param n The size of the matrices
 * @param A The A matrix in dense format
 * @param B The B matrix in dense format
 * @param eigenvalues The eigenvalues (output)
 * @param eigenvectors The eigenvectors (output)
 * @param num_eigenvalues The number of eigenvalues to compute
 * @return hipError_t The CUDA error code
 */
hipError_t solve_generalized_eigenvalue_problem(
    int n,
    thrust::complex<double>* A,
    thrust::complex<double>* B,
    double* eigenvalues,
    thrust::complex<double>* eigenvectors,
    int num_eigenvalues) {

    // Create cuSOLVER handle
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t status = hipsolverDnCreate(&cusolverH);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return hipErrorInvalidValue;
    }

    // Workspace size query
    int lwork = 0;
    status = hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_LOWER,
        n,
        reinterpret_cast<hipDoubleComplex*>(A),
        n,
        reinterpret_cast<hipDoubleComplex*>(B),
        n,
        eigenvalues,
        &lwork
    );

    if (status != HIPSOLVER_STATUS_SUCCESS) {
        hipsolverDnDestroy(cusolverH);
        return hipErrorInvalidValue;
    }

    // Allocate workspace
    hipDoubleComplex* work = NULL;
    hipError_t cuda_status = hipMalloc((void**)&work, sizeof(hipDoubleComplex) * lwork);
    if (cuda_status != hipSuccess) {
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }

    // Allocate device memory for info
    int* devInfo = NULL;
    cuda_status = hipMalloc((void**)&devInfo, sizeof(int));
    if (cuda_status != hipSuccess) {
        hipFree(work);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }

    // Solve generalized eigenvalue problem
    status = hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_LOWER,
        n,
        reinterpret_cast<hipDoubleComplex*>(A),
        n,
        reinterpret_cast<hipDoubleComplex*>(B),
        n,
        eigenvalues,
        work,
        lwork,
        devInfo
    );

    if (status != HIPSOLVER_STATUS_SUCCESS) {
        hipFree(devInfo);
        hipFree(work);
        hipsolverDnDestroy(cusolverH);
        return hipErrorInvalidValue;
    }

    // Check for errors
    int info = 0;
    cuda_status = hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess || info != 0) {
        hipFree(devInfo);
        hipFree(work);
        hipsolverDnDestroy(cusolverH);
        return hipErrorInvalidValue;
    }

    // Copy eigenvectors
    cuda_status = hipMemcpy(eigenvectors, A, sizeof(thrust::complex<double>) * n * n, hipMemcpyDeviceToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(devInfo);
        hipFree(work);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }

    // Clean up
    hipFree(devInfo);
    hipFree(work);
    hipsolverDnDestroy(cusolverH);

    return hipSuccess;
}

/**
 * @brief Solves the generalized eigenvalue problem Ax = λBx for sparse matrices.
 *
 * @param n The size of the matrices
 * @param nnz_A The number of non-zero elements in A
 * @param nnz_B The number of non-zero elements in B
 * @param csrRowPtrA The CSR row pointer array for A
 * @param csrColIndA The CSR column index array for A
 * @param csrValA The CSR value array for A
 * @param csrRowPtrB The CSR row pointer array for B
 * @param csrColIndB The CSR column index array for B
 * @param csrValB The CSR value array for B
 * @param eigenvalues The eigenvalues (output)
 * @param eigenvectors The eigenvectors (output)
 * @param num_eigenvalues The number of eigenvalues to compute
 * @return hipError_t The CUDA error code
 */
extern "C" hipError_t solve_sparse_generalized_eigenvalue_problem(
    int n,
    int nnz_A,
    int nnz_B,
    const int* csrRowPtrA,
    const int* csrColIndA,
    const std::complex<double>* csrValA,
    const int* csrRowPtrB,
    const int* csrColIndB,
    const std::complex<double>* csrValB,
    double* eigenvalues,
    std::complex<double>* eigenvectors,
    int num_eigenvalues) {

    // Create cuSPARSE handle
    hipsparseHandle_t cusparseH = NULL;
    hipsparseStatus_t status = hipsparseCreate(&cusparseH);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        return hipErrorInvalidValue;
    }

    // Allocate device memory for CSR matrices
    int* d_csrRowPtrA = NULL;
    int* d_csrColIndA = NULL;
    thrust::complex<double>* d_csrValA = NULL;

    int* d_csrRowPtrB = NULL;
    int* d_csrColIndB = NULL;
    thrust::complex<double>* d_csrValB = NULL;

    hipError_t cuda_status = hipMalloc((void**)&d_csrRowPtrA, sizeof(int) * (n + 1));
    if (cuda_status != hipSuccess) {
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&d_csrColIndA, sizeof(int) * nnz_A);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&d_csrValA, sizeof(thrust::complex<double>) * nnz_A);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&d_csrRowPtrB, sizeof(int) * (n + 1));
    if (cuda_status != hipSuccess) {
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&d_csrColIndB, sizeof(int) * nnz_B);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&d_csrValB, sizeof(thrust::complex<double>) * nnz_B);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    // Copy CSR matrices to device
    cuda_status = hipMemcpy(d_csrRowPtrA, csrRowPtrA, sizeof(int) * (n + 1), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMemcpy(d_csrColIndA, csrColIndA, sizeof(int) * nnz_A, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMemcpy(d_csrValA, csrValA, sizeof(thrust::complex<double>) * nnz_A, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMemcpy(d_csrRowPtrB, csrRowPtrB, sizeof(int) * (n + 1), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMemcpy(d_csrColIndB, csrColIndB, sizeof(int) * nnz_B, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMemcpy(d_csrValB, csrValB, sizeof(thrust::complex<double>) * nnz_B, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    // Allocate device memory for dense matrices
    thrust::complex<double>* d_A = NULL;
    thrust::complex<double>* d_B = NULL;

    cuda_status = hipMalloc((void**)&d_A, sizeof(thrust::complex<double>) * n * n);
    if (cuda_status != hipSuccess) {
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&d_B, sizeof(thrust::complex<double>) * n * n);
    if (cuda_status != hipSuccess) {
        hipFree(d_A);
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    // Convert CSR to dense
    cuda_status = csr2dense(cusparseH, n, n, nnz_A, d_csrRowPtrA, d_csrColIndA, d_csrValA, d_A);
    if (cuda_status != hipSuccess) {
        hipFree(d_B);
        hipFree(d_A);
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = csr2dense(cusparseH, n, n, nnz_B, d_csrRowPtrB, d_csrColIndB, d_csrValB, d_B);
    if (cuda_status != hipSuccess) {
        hipFree(d_B);
        hipFree(d_A);
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    // Allocate device memory for eigenvalues and eigenvectors
    double* d_eigenvalues = NULL;
    thrust::complex<double>* d_eigenvectors = NULL;

    cuda_status = hipMalloc((void**)&d_eigenvalues, sizeof(double) * n);
    if (cuda_status != hipSuccess) {
        hipFree(d_B);
        hipFree(d_A);
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&d_eigenvectors, sizeof(thrust::complex<double>) * n * n);
    if (cuda_status != hipSuccess) {
        hipFree(d_eigenvalues);
        hipFree(d_B);
        hipFree(d_A);
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    // Solve generalized eigenvalue problem
    cuda_status = solve_generalized_eigenvalue_problem(n, d_A, d_B, d_eigenvalues, d_eigenvectors, num_eigenvalues);
    if (cuda_status != hipSuccess) {
        hipFree(d_eigenvectors);
        hipFree(d_eigenvalues);
        hipFree(d_B);
        hipFree(d_A);
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    // Copy results back to host
    cuda_status = hipMemcpy(eigenvalues, d_eigenvalues, sizeof(double) * num_eigenvalues, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        hipFree(d_eigenvectors);
        hipFree(d_eigenvalues);
        hipFree(d_B);
        hipFree(d_A);
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    cuda_status = hipMemcpy(eigenvectors, d_eigenvectors, sizeof(thrust::complex<double>) * n * num_eigenvalues, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        hipFree(d_eigenvectors);
        hipFree(d_eigenvalues);
        hipFree(d_B);
        hipFree(d_A);
        hipFree(d_csrValB);
        hipFree(d_csrColIndB);
        hipFree(d_csrRowPtrB);
        hipFree(d_csrValA);
        hipFree(d_csrColIndA);
        hipFree(d_csrRowPtrA);
        hipsparseDestroy(cusparseH);
        return cuda_status;
    }

    // Clean up
    hipFree(d_eigenvectors);
    hipFree(d_eigenvalues);
    hipFree(d_B);
    hipFree(d_A);
    hipFree(d_csrValB);
    hipFree(d_csrColIndB);
    hipFree(d_csrRowPtrB);
    hipFree(d_csrValA);
    hipFree(d_csrColIndA);
    hipFree(d_csrRowPtrA);
    hipsparseDestroy(cusparseH);

    return hipSuccess;
}
