/**
 * @file gpu_cusolver.cu
 * @brief CUDA implementation of eigenvalue solver using cuSOLVER.
 *
 * This file contains the CUDA implementation of the eigenvalue solver using
 * cuSOLVER for the Schrödinger equation.
 *
 * Author: Dr. Mazharuddin Mohammed
 */

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <complex>
#include <vector>
#include <iostream>
#include <Eigen/Dense>
#include <Eigen/Sparse>

#include "gpu_accelerator.h"

/**
 * @brief Solves the generalized eigenvalue problem Ax = λBx using cuSOLVER.
 *
 * This function solves the generalized eigenvalue problem Ax = λBx using
 * cuSOLVER. It computes the lowest `num_eigenvalues` eigenvalues and
 * corresponding eigenvectors.
 *
 * @param A The A matrix
 * @param B The B matrix
 * @param num_eigenvalues The number of eigenvalues to compute
 * @param eigenvalues The eigenvalues (output)
 * @param eigenvectors The eigenvectors (output)
 * @return hipError_t The CUDA error code
 */
hipError_t solve_generalized_eigenvalue_problem_cusolver(
    const Eigen::MatrixXcd& A,
    const Eigen::MatrixXcd& B,
    int num_eigenvalues,
    std::vector<double>& eigenvalues,
    std::vector<Eigen::VectorXd>& eigenvectors) {
    
    // Get matrix dimensions
    int n = A.rows();
    
    // Create cuSOLVER handle
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t status = hipsolverDnCreate(&cusolverH);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return hipErrorInvalidValue;
    }
    
    // Allocate device memory for matrices
    hipDoubleComplex* d_A = NULL;
    hipDoubleComplex* d_B = NULL;
    double* d_W = NULL;
    
    hipError_t cuda_status = hipMalloc((void**)&d_A, sizeof(hipDoubleComplex) * n * n);
    if (cuda_status != hipSuccess) {
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    cuda_status = hipMalloc((void**)&d_B, sizeof(hipDoubleComplex) * n * n);
    if (cuda_status != hipSuccess) {
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    cuda_status = hipMalloc((void**)&d_W, sizeof(double) * n);
    if (cuda_status != hipSuccess) {
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    // Copy matrices to device
    cuda_status = hipMemcpy(d_A, A.data(), sizeof(hipDoubleComplex) * n * n, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    cuda_status = hipMemcpy(d_B, B.data(), sizeof(hipDoubleComplex) * n * n, hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    // Workspace size query
    int lwork = 0;
    status = hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_LOWER,
        n,
        d_A,
        n,
        d_B,
        n,
        d_W,
        &lwork
    );
    
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return hipErrorInvalidValue;
    }
    
    // Allocate workspace
    hipDoubleComplex* d_work = NULL;
    cuda_status = hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork);
    if (cuda_status != hipSuccess) {
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    // Allocate device memory for info
    int* d_info = NULL;
    cuda_status = hipMalloc((void**)&d_info, sizeof(int));
    if (cuda_status != hipSuccess) {
        hipFree(d_work);
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    // Solve generalized eigenvalue problem
    status = hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_LOWER,
        n,
        d_A,
        n,
        d_B,
        n,
        d_W,
        d_work,
        lwork,
        d_info
    );
    
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        hipFree(d_info);
        hipFree(d_work);
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return hipErrorInvalidValue;
    }
    
    // Check for errors
    int info = 0;
    cuda_status = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess || info != 0) {
        hipFree(d_info);
        hipFree(d_work);
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return hipErrorInvalidValue;
    }
    
    // Copy eigenvalues to host
    std::vector<double> all_eigenvalues(n);
    cuda_status = hipMemcpy(all_eigenvalues.data(), d_W, sizeof(double) * n, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        hipFree(d_info);
        hipFree(d_work);
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    // Copy eigenvectors to host
    Eigen::MatrixXcd all_eigenvectors(n, n);
    cuda_status = hipMemcpy(all_eigenvectors.data(), d_A, sizeof(hipDoubleComplex) * n * n, hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        hipFree(d_info);
        hipFree(d_work);
        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipsolverDnDestroy(cusolverH);
        return cuda_status;
    }
    
    // Clean up
    hipFree(d_info);
    hipFree(d_work);
    hipFree(d_W);
    hipFree(d_B);
    hipFree(d_A);
    hipsolverDnDestroy(cusolverH);
    
    // Sort eigenvalues and eigenvectors
    std::vector<std::pair<double, int>> sorted_eigenvalues(n);
    for (int i = 0; i < n; ++i) {
        sorted_eigenvalues[i] = std::make_pair(all_eigenvalues[i], i);
    }
    std::sort(sorted_eigenvalues.begin(), sorted_eigenvalues.end());
    
    // Extract the lowest num_eigenvalues eigenvalues and eigenvectors
    eigenvalues.resize(num_eigenvalues);
    eigenvectors.resize(num_eigenvalues);
    
    for (int i = 0; i < num_eigenvalues; ++i) {
        eigenvalues[i] = sorted_eigenvalues[i].first;
        
        // Extract eigenvector
        Eigen::VectorXcd complex_eigenvector = all_eigenvectors.col(sorted_eigenvalues[i].second);
        
        // Convert to real
        eigenvectors[i] = complex_eigenvector.cwiseAbs();
    }
    
    return hipSuccess;
}

/**
 * @brief Implementation of the solve_eigen_cusolver method for the GPUAccelerator class.
 *
 * This function is the implementation of the solve_eigen_cusolver method for the
 * GPUAccelerator class. It solves the generalized eigenvalue problem Ax = λBx
 * using cuSOLVER.
 *
 * @param A The A matrix
 * @param B The B matrix
 * @param num_eigenvalues The number of eigenvalues to compute
 * @param eigenvalues The eigenvalues (output)
 * @param eigenvectors The eigenvectors (output)
 */
void GPUAccelerator::solve_eigen_cusolver(
    const Eigen::MatrixXcd& A,
    const Eigen::MatrixXcd& B,
    int num_eigenvalues,
    std::vector<std::complex<double>>& eigenvalues,
    std::vector<Eigen::VectorXd>& eigenvectors) {
    
    // Solve the generalized eigenvalue problem
    std::vector<double> real_eigenvalues;
    hipError_t cuda_status = solve_generalized_eigenvalue_problem_cusolver(
        A, B, num_eigenvalues, real_eigenvalues, eigenvectors
    );
    
    if (cuda_status != hipSuccess) {
        throw std::runtime_error("Failed to solve generalized eigenvalue problem using cuSOLVER: " +
                                std::string(hipGetErrorString(cuda_status)));
    }
    
    // Convert eigenvalues to complex
    eigenvalues.resize(num_eigenvalues);
    for (int i = 0; i < num_eigenvalues; ++i) {
        eigenvalues[i] = std::complex<double>(real_eigenvalues[i], 0.0);
    }
}
